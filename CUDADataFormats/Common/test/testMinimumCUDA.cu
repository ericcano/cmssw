#include <hip/hip_runtime.h>

#include <iostream>

int main(void) {
  int devices = 0;
  hipError_t st = hipGetDeviceCount(&devices);
  std::cout << "st= " << hipGetErrorString(st) << " count=" << devices << std::endl;  
  return 0;
}